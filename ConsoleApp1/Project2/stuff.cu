#include <stdio.h>
#include <hip/hip_runtime.h>

const int N = 11;
const int blocksize = 11;

__global__
void hello(char * ar)
{
	int i = threadIdx.x;
	ar[i] = ar[i] + 1;
}

int main()
{
	char a[] = { 'G', 'd', 'k', 'k', 'n', (char)31, 'v', 'n', 'q', 'k', 'c', 0 };
	char *ad;
	const int csize = N * sizeof(char);

	printf("%s\n", a);

	int res = hipMalloc((void**)&ad, csize);
	res = hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello<<<dimGrid, dimBlock>>>(ad);
	res = hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	res = hipFree(ad);

	printf("%s\n", a);
	return 0;
}